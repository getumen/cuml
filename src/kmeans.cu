#include "cuda_utils.h"
#include "handle_utils.h"
#include "preprocessor.h"
#include "stream_allocator.h"
#include "device_vector_utils.h"
#include "cuml4c/device_vector.h"
#include "cuml4c/kmeans.h"

#include <thrust/device_vector.h>
#include <cuml/cluster/kmeans.hpp>

#include <memory>

// TODO: support initMethod == Array
__host__ int KmeansFit(
    DeviceVectorHandleFloat device_x,
    int num_row,
    int num_col,
    DeviceVectorHandleFloat device_sample_weight,
    int k,
    int max_iters,
    double tol,
    int init_method,
    int metric,
    int seed,
    int verbosity,
    DeviceVectorHandleInt device_labels,
    DeviceVectorHandleFloat device_centroids,
    float *inertia,
    int *n_iter)
{

    auto d_x = static_cast<cuml4c::DeviceVector<float> *>(device_x);
    auto d_sample_weight = static_cast<cuml4c::DeviceVector<float> *>(device_sample_weight);
    auto d_labels = static_cast<cuml4c::DeviceVector<int> *>(device_labels);
    auto d_centroids = static_cast<cuml4c::DeviceVector<float> *>(device_centroids);

    ML::kmeans::KMeansParams params;
    params.n_clusters = k;
    params.max_iter = max_iters;
    if (tol > 0)
    {
        params.tol = tol;
        params.inertia_check = true;
    }

    params.init = static_cast<ML::kmeans::KMeansParams::InitMethod>(init_method);
    params.seed = seed;
    params.verbosity = verbosity;
    params.metric = metric;

    auto stream_view = cuml4c::stream_allocator::getOrCreateStream();
    raft::handle_t handle;
    cuml4c::handle_utils::initializeHandle(handle, stream_view.value());

    if (device_sample_weight == nullptr)
    {
        ML::kmeans::fit_predict(
            handle,
            params,
            d_x->vector->data().get(),
            num_row,
            num_col,
            nullptr,
            d_centroids->vector->data().get(),
            d_labels->vector->data().get(),
            *inertia,
            *n_iter);
    }
    else
    {

        ML::kmeans::fit_predict(
            handle,
            params,
            d_x->vector->data().get(),
            num_row,
            num_col,
            d_sample_weight->vector->data().get(),
            d_centroids->vector->data().get(),
            d_labels->vector->data().get(),
            *inertia,
            *n_iter);
    }

    return 0;
}
