#include "async_utils.cuh"
#include "cuda_utils.h"
#include "handle_utils.h"
#include "preprocessor.h"
#include "stream_allocator.h"
#include "cuml4c/kmeans.h"

#include <thrust/async/copy.h>
#include <thrust/device_vector.h>
#include <cuml/cluster/kmeans.hpp>

#include <memory>

__host__ int KmeansFit(
    const float *x,
    int num_row,
    int num_col,
    const float *sample_weight,
    int k,
    int max_iters,
    double tol,
    int init_method,
    int metric,
    int seed,
    int verbosity,
    int *labels,
    float *centroids,
    float *inertia,
    int *n_iter)
{

    ML::kmeans::KMeansParams params;
    params.n_clusters = k;
    params.max_iter = max_iters;
    if (tol > 0)
    {
        params.tol = tol;
        params.inertia_check = true;
    }

    params.init = static_cast<ML::kmeans::KMeansParams::InitMethod>(init_method);
    params.seed = seed;
    params.verbosity = verbosity;
    params.metric = metric;

    auto stream_view = cuml4c::stream_allocator::getOrCreateStream();
    raft::handle_t handle;
    cuml4c::handle_utils::initializeHandle(handle, stream_view.value());

    // kmeans input data
    const auto value_length = num_row * num_col;

    auto const n_centroid_values = params.n_clusters * num_col;
    thrust::device_vector<float> d_src_data(value_length);
    // TODO: async copy
    thrust::copy(
        x,
        x + value_length,
        d_src_data.begin());

    thrust::device_vector<float> d_sample_weight(num_row);
    if (sample_weight != nullptr)
    {
        thrust::copy(
            sample_weight,
            sample_weight + num_row,
            d_sample_weight.begin());
    }
    else
    {
        thrust::fill(
            d_sample_weight.begin(),
            d_sample_weight.end(),
            1.0f);
    }

    // kmeans outputs
    thrust::device_vector<float> d_pred_centroids(n_centroid_values);
    if (params.init == ML::kmeans::KMeansParams::InitMethod::Array)
    {
        // TODO: async copy
        thrust::copy(
            centroids,
            centroids + n_centroid_values,
            d_pred_centroids.begin());
    }
    thrust::device_vector<int> d_pred_labels(num_row);

    ML::kmeans::fit_predict(
        handle,
        params,
        d_src_data.data().get(),
        num_row,
        num_col,
        d_sample_weight.data().get(),
        d_pred_centroids.data().get(),
        d_pred_labels.data().get(),
        *inertia,
        *n_iter);

    // TODO: async copy
    thrust::copy(
        d_pred_labels.begin(),
        d_pred_labels.end(),
        labels);

    // TODO: async copy
    thrust::copy(
        d_pred_centroids.begin(),
        d_pred_centroids.end(),
        centroids);

    return 0;
}
