#include "cuml4c/kmeans.h"

#include <thrust/copy.h>
#include <raft/core/handle.hpp>
#include <rmm/device_uvector.hpp>
#include <cuml/cluster/kmeans.hpp>

#include <memory>

__host__ int KmeansFit(
    const float *x,
    int num_row,
    int num_col,
    int k,
    int max_iters,
    double tol,
    int init_method,
    int metric,
    int seed,
    int verbosity,
    int *labels,
    float *centroids,
    float *inertia,
    int *n_iter)
{

    auto handle = std::make_shared<raft::handle_t>();

    auto d_x = rmm::device_uvector<float>(
        num_col * num_row,
        handle->get_stream());

    raft::update_device(d_x.data(),
                        x,
                        num_col * num_row,
                        handle->get_stream());

    auto d_labels = rmm::device_uvector<int>(
        num_row,
        handle->get_stream());

    auto d_centroids = rmm::device_uvector<float>(
        k * num_col,
        handle->get_stream());

    ML::kmeans::KMeansParams params;
    params.n_clusters = k;
    params.max_iter = max_iters;
    if (tol > 0)
    {
        params.tol = tol;
        params.inertia_check = true;
    }

    params.init = static_cast<ML::kmeans::KMeansParams::InitMethod>(init_method);
    params.verbosity = verbosity;
    params.metric = static_cast<raft::distance::DistanceType>(metric);

    ML::kmeans::fit_predict(
        *handle,
        params,
        d_x.begin(),
        num_row,
        num_col,
        nullptr,
        d_centroids.begin(),
        d_labels.begin(),
        *inertia,
        *n_iter);

    raft::update_host(labels,
                      d_labels.begin(),
                      d_labels.size(),
                      handle->get_stream());

    raft::update_host(centroids,
                      d_centroids.begin(),
                      d_centroids.size(),
                      handle->get_stream());

    handle->get_stream().synchronize();

    return 0;
}
