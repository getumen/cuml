#include "cuda_utils.h"

namespace cuml4c
{

    __host__ int currentDevice()
    {
        int dev_id;
        CUDA_RT_CALL(hipGetDevice(&dev_id));
        return dev_id;
    }

} // namespace cuml4c
