#include "cuda_utils.h"
#include "handle_utils.h"
#include "preprocessor.h"
#include "stream_allocator.h"
#include "device_vector_utils.h"
#include "cuml4c/device_vector.h"
#include "cuml4c/dbscan.h"

#include <thrust/device_vector.h>
#include <cuml/cluster/dbscan.hpp>

#include <memory>

__host__ int DbscanFit(
    DeviceVectorHandleFloat device_x,
    size_t num_row,
    size_t num_col,
    int min_pts,
    double eps,
    int metric,
    size_t max_bytes_per_batch,
    int verbosity,
    DeviceVectorHandleInt device_labels)
{

    auto d_x = static_cast<cuml4c::DeviceVector<float> *>(device_x);
    auto d_labels = static_cast<cuml4c::DeviceVector<int> *>(device_labels);

    auto stream_view = cuml4c::stream_allocator::getOrCreateStream();
    raft::handle_t handle;
    cuml4c::handle_utils::initializeHandle(handle, stream_view.value());

    ML::Dbscan::fit(handle,
                    /*input=*/d_x->vector->data().get(),
                    /*n_rows=*/num_row,
                    /*n_cols=*/num_col,
                    eps,
                    min_pts,
                    /*metric=*/static_cast<raft::distance::DistanceType>(metric),
                    /*labels=*/d_labels->vector->data().get(),
                    /*core_sample_indices=*/nullptr,
                    max_bytes_per_batch,
                    /*verbosity=*/verbosity,
                    /*opg=*/false);

    return 0;
}
