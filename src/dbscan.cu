#include "cuml4c/dbscan.h"
#include "device_resource_handle.cuh"

#include <thrust/copy.h>
#include <raft/core/handle.hpp>
#include <rmm/device_uvector.hpp>
#include <cuml/cluster/dbscan.hpp>

#include <memory>

__host__ int DbscanFit(
    const DeviceResourceHandle handle,
    const float *x,
    size_t num_row,
    size_t num_col,
    int min_pts,
    double eps,
    int metric,
    size_t max_bytes_per_batch,
    int verbosity,
    int *labels)
{
    auto handle_p = static_cast<cuml4c::DeviceResource *>(handle);

    auto d_x = rmm::device_uvector<float>(
        num_col * num_row,
        handle_p->handle->get_stream());

    raft::update_device(d_x.data(),
                        x,
                        num_col * num_row,
                        handle_p->handle->get_stream());

    auto d_labels = rmm::device_uvector<int>(
        num_row,
        handle_p->handle->get_stream());

    ML::Dbscan::fit(*handle_p->handle,
                    /*input=*/d_x.begin(),
                    /*n_rows=*/num_row,
                    /*n_cols=*/num_col,
                    eps,
                    min_pts,
                    /*metric=*/static_cast<raft::distance::DistanceType>(metric),
                    /*labels=*/d_labels.begin(),
                    /*core_sample_indices=*/nullptr,
                    max_bytes_per_batch,
                    /*verbosity=*/verbosity,
                    /*opg=*/false);

    raft::update_host(labels,
                      d_labels.begin(),
                      d_labels.size(),
                      handle_p->handle->get_stream());

    handle_p->handle->get_stream().synchronize();

    return 0;
}
