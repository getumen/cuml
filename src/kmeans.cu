#include "cuml4c/kmeans.h"
#include "device_resource_handle.cuh"

#include <raft/core/handle.hpp>
#include <rmm/device_uvector.hpp>
#include <cuml/cluster/kmeans.hpp>

#include <memory>

__host__ int
KmeansFit(
    const DeviceResourceHandle handle,
    const float *x,
    int num_row,
    int num_col,
    int k,
    int max_iters,
    double tol,
    int init_method,
    int metric,
    int seed,
    int verbosity,
    int *labels,
    float *centroids,
    float *inertia,
    int *n_iter)
{
    auto handle_p = static_cast<cuml4c::DeviceResource *>(handle);

    auto d_x = rmm::device_uvector<float>(
        num_col * num_row,
        handle_p->handle->get_stream());

    raft::update_device(d_x.data(),
                        x,
                        num_col * num_row,
                        handle_p->handle->get_stream());

    auto d_labels = rmm::device_uvector<int>(
        num_row,
        handle_p->handle->get_stream());

    auto d_centroids = rmm::device_uvector<float>(
        k * num_col,
        handle_p->handle->get_stream());

    ML::kmeans::KMeansParams params;
    params.n_clusters = k;
    params.max_iter = max_iters;
    if (tol > 0)
    {
        params.tol = tol;
        params.inertia_check = true;
    }

    params.init = static_cast<ML::kmeans::KMeansParams::InitMethod>(init_method);
    params.verbosity = verbosity;
    params.metric = static_cast<raft::distance::DistanceType>(metric);

    ML::kmeans::fit_predict(
        *handle_p->handle,
        params,
        d_x.begin(),
        num_row,
        num_col,
        nullptr,
        d_centroids.begin(),
        d_labels.begin(),
        *inertia,
        *n_iter);

    raft::update_host(labels,
                      d_labels.begin(),
                      d_labels.size(),
                      handle_p->handle->get_stream());

    raft::update_host(centroids,
                      d_centroids.begin(),
                      d_centroids.size(),
                      handle_p->handle->get_stream());

    handle_p->handle->sync_stream();

    return 0;
}
