#include "cuml4c/dbscan.h"

#include <thrust/copy.h>
#include <raft/core/handle.hpp>
#include <rmm/device_uvector.hpp>
#include <cuml/cluster/dbscan.hpp>

#include <memory>

__host__ int DbscanFit(
    const float *x,
    size_t num_row,
    size_t num_col,
    int min_pts,
    double eps,
    int metric,
    size_t max_bytes_per_batch,
    int verbosity,
    int *labels)
{
    auto handle = std::make_unique<raft::handle_t>();

    auto d_x = rmm::device_uvector<float>(
        num_col * num_row,
        handle->get_stream());

    raft::update_device(d_x.data(),
                        x,
                        num_col * num_row,
                        handle->get_stream());

    auto d_labels = rmm::device_uvector<int>(
        num_row,
        handle->get_stream());

    ML::Dbscan::fit(*handle,
                    /*input=*/d_x.begin(),
                    /*n_rows=*/num_row,
                    /*n_cols=*/num_col,
                    eps,
                    min_pts,
                    /*metric=*/static_cast<raft::distance::DistanceType>(metric),
                    /*labels=*/d_labels.begin(),
                    /*core_sample_indices=*/nullptr,
                    max_bytes_per_batch,
                    /*verbosity=*/verbosity,
                    /*opg=*/false);

    raft::update_host(labels,
                      d_labels.begin(),
                      d_labels.size(),
                      handle->get_stream());

    handle->get_stream().synchronize();

    return 0;
}
