#include "async_utils.cuh"
#include "cuda_utils.h"
#include "handle_utils.h"
#include "preprocessor.h"
#include "stream_allocator.h"
#include "cuml4c/agglomerative_clustering.h"

#include <thrust/async/copy.h>
#include <thrust/device_vector.h>
#include <cuml/cluster/linkage.hpp>

#include <memory>

__host__ int AgglomerativeClusteringFit(
    const float *x,
    size_t num_row,
    size_t num_col,
    bool pairwise_conn,
    int metric,
    int n_neighbors,
    int init_n_clusters,
    int *n_clusters,
    int *labels,
    int *children)
{

    const auto value_length = num_row * num_col;

    auto stream_view = cuml4c::stream_allocator::getOrCreateStream();
    raft::handle_t handle;
    cuml4c::handle_utils::initializeHandle(handle, stream_view.value());

    // single-linkage hierarchical clustering input
    thrust::device_vector<float> d_x(value_length);
    thrust::copy(x, x + value_length, d_x.begin());

    // single-linkage hierarchical clustering output
    auto out = std::make_unique<raft::hierarchy::linkage_output<int, float>>();
    thrust::device_vector<int> d_labels(num_row);
    thrust::device_vector<int> d_children((num_row - 1) * 2);
    out->labels = d_labels.data().get();
    out->children = d_children.data().get();

    if (pairwise_conn)
    {
        ML::single_linkage_pairwise(
            handle,
            /*X=*/d_x.data().get(),
            /*m=*/num_row,
            /*n=*/num_col,
            /*out=*/out.get(),
            /*metric=*/static_cast<raft::distance::DistanceType>(metric),
            init_n_clusters);
    }
    else
    {
        ML::single_linkage_neighbors(
            handle,
            /*X=*/d_x.data().get(),
            /*m=*/num_row,
            /*n=*/num_col,
            /*out=*/out.get(),
            /*metric=*/static_cast<raft::distance::DistanceType>(metric),
            /*c=*/n_neighbors,
            init_n_clusters);
    }

    thrust::copy(
        d_labels.begin(),
        d_labels.end(),
        labels);
    thrust::copy(
        d_children.begin(),
        d_children.end(),
        children);

    *n_clusters = out->n_clusters;

    return 0;
}
