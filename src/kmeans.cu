#include "cuda_utils.h"
#include "handle_utils.h"
#include "preprocessor.h"
#include "stream_allocator.h"
#include "device_vector_utils.h"
#include "cuml4c/device_vector.h"
#include "cuml4c/kmeans.h"

#include <thrust/device_vector.h>
#include <cuml/cluster/kmeans.hpp>

#include <memory>

// TODO: support initMethod == Array
__host__ int KmeansFit(
    DeviceVectorHandleFloat device_x,
    int num_row,
    int num_col,
    DeviceVectorHandleFloat device_sample_weight,
    int k,
    int max_iters,
    double tol,
    int init_method,
    int metric,
    int seed,
    int verbosity,
    DeviceVectorHandleInt *device_labels,
    DeviceVectorHandleFloat *device_centroids,
    float *inertia,
    int *n_iter)
{

    auto d_x = static_cast<cuml4c::DeviceVector<float> *>(device_x);

    ML::kmeans::KMeansParams params;
    params.n_clusters = k;
    params.max_iter = max_iters;
    if (tol > 0)
    {
        params.tol = tol;
        params.inertia_check = true;
    }

    params.init = static_cast<ML::kmeans::KMeansParams::InitMethod>(init_method);
    params.seed = seed;
    params.verbosity = verbosity;
    params.metric = metric;

    auto stream_view = cuml4c::stream_allocator::getOrCreateStream();
    raft::handle_t handle;
    cuml4c::handle_utils::initializeHandle(handle, stream_view.value());

    auto const n_centroid_values = params.n_clusters * num_col;
    auto d_centroids = std::make_unique<thrust::device_vector<float>>(n_centroid_values);

    auto d_labels = std::make_unique<thrust::device_vector<int>>(num_row);

    if (device_sample_weight == nullptr)
    {
        ML::kmeans::fit_predict(
            handle,
            params,
            d_x->vector->data().get(),
            num_row,
            num_col,
            nullptr,
            d_centroids->data().get(),
            d_labels->data().get(),
            *inertia,
            *n_iter);
    }
    else
    {
        auto d_sample_weight = static_cast<cuml4c::DeviceVector<float> *>(device_sample_weight);

        ML::kmeans::fit_predict(
            handle,
            params,
            d_x->vector->data().get(),
            num_row,
            num_col,
            d_sample_weight->vector->data().get(),
            d_centroids->data().get(),
            d_labels->data().get(),
            *inertia,
            *n_iter);
    }

    auto p_labels = std::make_unique<cuml4c::DeviceVector<int>>(std::move(d_labels));
    *device_labels = static_cast<DeviceVectorHandleInt>(p_labels.release());

    auto p_centroids = std::make_unique<cuml4c::DeviceVector<float>>(std::move(d_centroids));
    *device_centroids = static_cast<DeviceVectorHandleFloat>(p_centroids.release());

    return 0;
}
