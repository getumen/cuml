#include "cuda_utils.h"
#include "handle_utils.h"
#include "preprocessor.h"
#include "stream_allocator.h"
#include "device_vector_utils.h"
#include "cuml4c/device_vector.h"
#include "cuml4c/agglomerative_clustering.h"

#include <thrust/device_vector.h>
#include <cuml/cluster/linkage.hpp>

#include <memory>

__host__ int AgglomerativeClusteringFit(
    DeviceVectorHandleFloat device_x,
    size_t num_row,
    size_t num_col,
    bool pairwise_conn,
    int metric,
    int n_neighbors,
    int init_n_clusters,
    int *n_clusters,
    DeviceVectorHandleInt *device_labels,
    DeviceVectorHandleInt *device_children)
{
    auto d_x = static_cast<cuml4c::DeviceVector<float> *>(device_x);

    auto d_labels = std::make_unique<thrust::device_vector<int>>(num_row);
    auto d_children = std::make_unique<thrust::device_vector<int>>((num_row - 1) * 2);

    auto stream_view = cuml4c::stream_allocator::getOrCreateStream();
    raft::handle_t handle;
    cuml4c::handle_utils::initializeHandle(handle, stream_view.value());

    // single-linkage hierarchical clustering output
    auto out = std::make_unique<raft::hierarchy::linkage_output<int, float>>();
    out->labels = d_labels->data().get();
    out->children = d_children->data().get();

    if (pairwise_conn)
    {
        ML::single_linkage_pairwise(
            handle,
            /*X=*/d_x->vector->data().get(),
            /*m=*/num_row,
            /*n=*/num_col,
            /*out=*/out.get(),
            /*metric=*/static_cast<raft::distance::DistanceType>(metric),
            init_n_clusters);
    }
    else
    {
        ML::single_linkage_neighbors(
            handle,
            /*X=*/d_x->vector->data().get(),
            /*m=*/num_row,
            /*n=*/num_col,
            /*out=*/out.get(),
            /*metric=*/static_cast<raft::distance::DistanceType>(metric),
            /*c=*/n_neighbors,
            init_n_clusters);
    }

    auto p_labels = std::make_unique<cuml4c::DeviceVector<int>>(std::move(d_labels));
    *device_labels = static_cast<DeviceVectorHandleInt>(p_labels.release());
    auto p_children = std::make_unique<cuml4c::DeviceVector<int>>(std::move(d_children));
    *device_children = static_cast<DeviceVectorHandleInt>(p_children.release());

    *n_clusters = out->n_clusters;

    return 0;
}
