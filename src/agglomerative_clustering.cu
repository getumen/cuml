#include "cuda_utils.h"
#include "handle_utils.h"
#include "preprocessor.h"
#include "stream_allocator.h"
#include "device_vector_utils.h"
#include "cuml4c/device_vector.h"
#include "cuml4c/agglomerative_clustering.h"

#include <thrust/device_vector.h>
#include <cuml/cluster/linkage.hpp>

#include <memory>

__host__ int AgglomerativeClusteringFit(
    DeviceVectorHandleFloat device_x,
    size_t num_row,
    size_t num_col,
    bool pairwise_conn,
    int metric,
    int n_neighbors,
    int init_n_clusters,
    int *n_clusters,
    DeviceVectorHandleInt device_labels,
    DeviceVectorHandleInt device_children)
{
    auto d_x = static_cast<cuml4c::DeviceVector<float> *>(device_x);

    auto d_labels = static_cast<cuml4c::DeviceVector<int> *>(device_labels);
    auto d_children = static_cast<cuml4c::DeviceVector<int> *>(device_children);

    auto stream_view = cuml4c::stream_allocator::getOrCreateStream();
    raft::handle_t handle;
    cuml4c::handle_utils::initializeHandle(handle, stream_view.value());

    // single-linkage hierarchical clustering output
    auto out = std::make_unique<raft::hierarchy::linkage_output<int, float>>();
    out->labels = d_labels->vector->data().get();
    out->children = d_children->vector->data().get();

    if (pairwise_conn)
    {
        ML::single_linkage_pairwise(
            handle,
            /*X=*/d_x->vector->data().get(),
            /*m=*/num_row,
            /*n=*/num_col,
            /*out=*/out.get(),
            /*metric=*/static_cast<raft::distance::DistanceType>(metric),
            init_n_clusters);
    }
    else
    {
        ML::single_linkage_neighbors(
            handle,
            /*X=*/d_x->vector->data().get(),
            /*m=*/num_row,
            /*n=*/num_col,
            /*out=*/out.get(),
            /*metric=*/static_cast<raft::distance::DistanceType>(metric),
            /*c=*/n_neighbors,
            init_n_clusters);
    }
    *n_clusters = out->n_clusters;

    return 0;
}
