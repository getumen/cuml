#include "cuml4c/fil.h"
#include "device_resource_handle.cuh"

#include <rmm/device_uvector.hpp>
#include <raft/core/handle.hpp>
#include <raft/util/cudart_utils.hpp>
#include <treelite/c_api.h>
#include <cuml/fil/fil.h>

#include <memory>
#include <string>
#include <fstream>
#include <iterator>

namespace
{

  enum class ModelType
  {
    XGBoost,
    XGBoostJSON,
    LightGBM
  };

  struct FILModel
  {
    __host__ FILModel(std::unique_ptr<ML::fil::forest32_t> forest,
                      int const num_features)
        : forest_(std::move(forest)),
          numFeatures_(num_features) {}

    std::unique_ptr<ML::fil::forest32_t> forest_;
    int const numFeatures_;
  };

  __host__ int treeliteLoadModel(ModelType const model_type,
                                 char const *filename,
                                 TreeliteModelHandle *model_handle)
  {
    std::string json_config = "{\"allow_unknown_field\": True}";
    switch (model_type)
    {
    case ModelType::XGBoost:
      return TreeliteLoadXGBoostModel(filename, json_config.c_str(), model_handle);
    case ModelType::XGBoostJSON: {
      std::ifstream file(filename); // Replace with your file name
      if (!file.is_open()) {
          return -1;
      }
      std::string content((std::istreambuf_iterator<char>(file)),
                        std::istreambuf_iterator<char>());
      file.close(); 

      return TreeliteLoadXGBoostModelFromString(content.c_str(), content.length(), json_config.c_str(), model_handle);
    }
    case ModelType::LightGBM:
      return TreeliteLoadLightGBMModel(filename, json_config.c_str(), model_handle);
    }

    // unreachable
    return -1;
  }

} // namespace

__host__ int FILLoadModel(
    const DeviceResourceHandle handle,
    int model_type,
    const char *filename,
    int algo,
    bool classification,
    float threshold,
    int storage_type,
    int blocks_per_sm,
    int threads_per_tree,
    int n_items,
    FILModelHandle *out)
{
  auto handle_p = static_cast<cuml4c::DeviceResource *>(handle);

  TreeliteModelHandle model_handle;
  {
    auto const res = treeliteLoadModel(
        /*model_type=*/static_cast<ModelType>(model_type),
        /*filename=*/filename,
        &model_handle);
    if (res < 0)
    {
      return FIL_FAIL_TO_LOAD_MODEL;
    }
  }

  int num_features = 0;
  {
    auto res = TreeliteQueryNumFeature(model_handle, &num_features);
    if (res < 0)
    {
      return FIL_FAIL_TO_GET_NUM_FEATURE;
    }
  }

  ML::fil::treelite_params_t params;
  params.algo = static_cast<ML::fil::algo_t>(algo);
  params.output_class = classification;
  params.threshold = threshold;
  params.storage_type = static_cast<ML::fil::storage_type_t>(storage_type);
  params.blocks_per_sm = blocks_per_sm;
  params.output_class = classification;
  params.threads_per_tree = threads_per_tree;
  params.n_items = n_items;
  params.pforest_shape_str = nullptr;
  params.precision = ML::fil::precision_t::PRECISION_FLOAT32;

  ML::fil::forest_variant f;

  ML::fil::from_treelite(
      /*handle=*/*handle_p->handle,
      /*pforest=*/&f,
      /*model=*/model_handle,
      /*tl_params=*/&params);

  auto forest = std::make_unique<ML::fil::forest32_t>(std::move(std::get<ML::fil::forest32_t>(f)));

  auto model = std::make_unique<FILModel>(
      std::move(forest),
      num_features);

  *out = static_cast<FILModelHandle>(model.release());

  {
    auto res = TreeliteFreeModel(model_handle);
    if (res < 0)
    {
      return FIL_FAIL_TO_FREE_MODEL;
    }
  }

  return FIL_SUCCESS;
}

__host__ int FILFreeModel(
    const DeviceResourceHandle handle,
    FILModelHandle model)
{
  auto handle_p = static_cast<cuml4c::DeviceResource *>(handle);
  auto model_ptr = static_cast<FILModel const *>(model);
  ML::fil::free(*handle_p->handle, *model_ptr->forest_);
  delete model_ptr;
  return FIL_SUCCESS;
}

__host__ int FILPredict(
    const DeviceResourceHandle handle,
    FILModelHandle model,
    const float *x,
    size_t num_row,
    bool output_class_probabilities,
    float *preds)
{
  auto handle_p = static_cast<cuml4c::DeviceResource *>(handle);

  auto fil_model = static_cast<FILModel *>(model);

  auto d_x = rmm::device_uvector<float>(
      fil_model->numFeatures_ * num_row,
      handle_p->handle->get_stream());

  raft::update_device(d_x.data(),
                      x,
                      fil_model->numFeatures_ * num_row,
                      handle_p->handle->get_stream());

  auto pred_size = output_class_probabilities
                       ? 2 * num_row
                       : num_row;

  auto d_preds = rmm::device_uvector<float>(
      pred_size,
      handle_p->handle->get_stream());

  ML::fil::predict(/*h=*/*handle_p->handle,
                   /*f=*/*fil_model->forest_,
                   /*preds=*/d_preds.begin(),
                   /*data=*/d_x.begin(),
                   /*num_rows=*/num_row,
                   /*predict_proba=*/output_class_probabilities);

  raft::update_host(preds,
                    d_preds.begin(),
                    d_preds.size(),
                    handle_p->handle->get_stream());

  handle_p->handle->sync_stream();

  return FIL_SUCCESS;
}
