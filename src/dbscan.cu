#include "cuda_utils.h"
#include "handle_utils.h"
#include "preprocessor.h"
#include "stream_allocator.h"
#include "device_vector_utils.h"
#include "cuml4c/device_vector.h"
#include "cuml4c/dbscan.h"

#include <thrust/device_vector.h>
#include <cuml/cluster/dbscan.hpp>

#include <memory>

__host__ int DbscanFit(
    DeviceVectorHandleFloat device_x,
    size_t num_row,
    size_t num_col,
    int min_pts,
    double eps,
    int metric,
    size_t max_bytes_per_batch,
    int verbosity,
    DeviceVectorHandleInt *device_labels)
{

    auto d_x = static_cast<cuml4c::DeviceVector<float> *>(device_x);

    // dbscan output data
    auto d_labels = std::make_unique<thrust::device_vector<int>>(num_row);

    auto stream_view = cuml4c::stream_allocator::getOrCreateStream();
    raft::handle_t handle;
    cuml4c::handle_utils::initializeHandle(handle, stream_view.value());

    ML::Dbscan::fit(handle,
                    /*input=*/d_x->vector->data().get(),
                    /*n_rows=*/num_row,
                    /*n_cols=*/num_col,
                    eps,
                    min_pts,
                    /*metric=*/static_cast<raft::distance::DistanceType>(metric),
                    /*labels=*/d_labels->data().get(),
                    /*core_sample_indices=*/nullptr,
                    max_bytes_per_batch,
                    /*verbosity=*/verbosity,
                    /*opg=*/false);

    auto p_labels = std::make_unique<cuml4c::DeviceVector<int>>(std::move(d_labels));
    *device_labels = static_cast<DeviceVectorHandleInt>(p_labels.release());

    return 0;
}
