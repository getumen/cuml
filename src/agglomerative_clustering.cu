#include "cuml4c/agglomerative_clustering.h"
#include "device_resource_handle.cuh"

#include <thrust/copy.h>
#include <raft/core/handle.hpp>
#include <rmm/device_uvector.hpp>
#include <cuml/cluster/linkage.hpp>

#include <memory>

__host__ int AgglomerativeClusteringFit(
    const DeviceResourceHandle handle,
    const float *x,
    size_t num_row,
    size_t num_col,
    bool pairwise_conn,
    int metric,
    int n_neighbors,
    int init_n_clusters,
    int *n_clusters,
    int *labels,
    int *children)
{
    auto handle_p = static_cast<cuml4c::DeviceResource *>(handle);

    auto d_x = rmm::device_uvector<float>(
        num_col * num_row,
        handle_p->handle->get_stream());

    raft::update_device(d_x.data(),
                        x,
                        num_col * num_row,
                        handle_p->handle->get_stream());

    auto d_labels = rmm::device_uvector<int>(
        num_row,
        handle_p->handle->get_stream());

    auto d_children = rmm::device_uvector<int>(
        (num_row - 1) * 2,
        handle_p->handle->get_stream());

    // single-linkage hierarchical clustering output
    auto out = std::make_unique<raft::hierarchy::linkage_output<int>>();
    out->labels = d_labels.begin();
    out->children = d_children.begin();

    if (pairwise_conn)
    {
        ML::single_linkage_pairwise(
            *handle_p->handle,
            /*X=*/d_x.begin(),
            /*m=*/num_row,
            /*n=*/num_col,
            /*out=*/out.get(),
            /*metric=*/static_cast<raft::distance::DistanceType>(metric),
            init_n_clusters);
    }
    else
    {
        ML::single_linkage_neighbors(
            *handle_p->handle,
            /*X=*/d_x.begin(),
            /*m=*/num_row,
            /*n=*/num_col,
            /*out=*/out.get(),
            /*metric=*/static_cast<raft::distance::DistanceType>(metric),
            /*c=*/n_neighbors,
            init_n_clusters);
    }
    *n_clusters = out->n_clusters;

    raft::update_host(labels,
                      d_labels.begin(),
                      d_labels.size(),
                      handle_p->handle->get_stream());

    raft::update_host(children,
                      d_children.begin(),
                      d_children.size(),
                      handle_p->handle->get_stream());

    handle_p->handle->get_stream().synchronize();

    return 0;
}
