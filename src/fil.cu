#include "hip/hip_runtime.h"
#include "async_utils.cuh"
#include "cuda_utils.h"
#include "fil_utils.h"
#include "handle_utils.h"
#include "preprocessor.h"
#include "stream_allocator.h"
#include "treelite_utils.cuh"
#include "cuml4c/fil.h"

#include <cuml/fil/fil.h>
#include <thrust/async/copy.h>
#include <thrust/device_vector.h>
#include <treelite/c_api.h>

#include <memory>
#include <string>

namespace
{

  enum class ModelType
  {
    XGBoost,
    XGBoostJSON,
    LightGBM
  };

  struct FILModel
  {
    __host__ FILModel(std::unique_ptr<raft::handle_t> handle,
                      cuml4c::fil::forest_uptr forest,
                      size_t const num_classes,
                      size_t const num_features)
        : handle_(std::move(handle)), forest_(std::move(forest)),
          numClasses_(num_classes), numFeatures_(num_features) {}

    std::unique_ptr<raft::handle_t> const handle_;
    // NOTE: the destruction of `forest_` must precede the destruction of
    // `handle_`.
    cuml4c::fil::forest_uptr forest_;
    size_t const numClasses_;
    size_t const numFeatures_;
  };

  __host__ int treeliteLoadModel(ModelType const model_type, char const *filename,
                                 cuml4c::TreeliteHandle &tl_handle)
  {
    switch (model_type)
    {
    case ModelType::XGBoost:
      return TreeliteLoadXGBoostModel(filename, tl_handle.get());
    case ModelType::XGBoostJSON:
      return TreeliteLoadXGBoostJSON(filename, tl_handle.get());
    case ModelType::LightGBM:
      return TreeliteLoadLightGBMModel(filename, tl_handle.get());
    }

    // unreachable
    return -1;
  }

  /*
   * The 'ML::fil::treelite_params_t::threads_per_tree' and
   * 'ML::fil::treelite_params_t::n_items' parameters are only supported in
   * RAPIDS cuML 21.08 or above.
   */
  CUML4C_ASSIGN_IF_PRESENT(threads_per_tree)
  CUML4C_NOOP_IF_ABSENT(threads_per_tree)

  CUML4C_ASSIGN_IF_PRESENT(n_items)
  CUML4C_NOOP_IF_ABSENT(n_items)

} // namespace

__host__ int FILLoadModel(
    int model_type,
    const char *filename,
    int algo,
    bool classification,
    float threshold,
    int storage_type,
    int blocks_per_sm,
    int threads_per_tree,
    int n_items,
    FILModelHandle *out)
{

  cuml4c::TreeliteHandle tl_handle;
  {
    auto const rc = treeliteLoadModel(
        /*model_type=*/static_cast<ModelType>(model_type),
        /*filename=*/filename,
        tl_handle);
    if (rc < 0)
    {
      return -1;
    }
  }

  ML::fil::treelite_params_t params;
  params.algo = static_cast<ML::fil::algo_t>(algo);
  params.output_class = classification;
  params.threshold = threshold;
  params.storage_type = static_cast<ML::fil::storage_type_t>(storage_type);
  params.blocks_per_sm = blocks_per_sm;
  params.output_class = classification;
  set_threads_per_tree(params, threads_per_tree);
  set_n_items(params, n_items);
  params.pforest_shape_str = nullptr;

  auto stream_view = cuml4c::stream_allocator::getOrCreateStream();
  auto handle = std::make_unique<raft::handle_t>();
  cuml4c::handle_utils::initializeHandle(*handle, stream_view.value());

  auto forest = cuml4c::fil::make_forest(*handle, /*src=*/[&]
                                         {
    ML::fil::forest *f;
    ML::fil::from_treelite(/*handle=*/*handle, /*pforest=*/&f,
                           /*model=*/*tl_handle.get(),
                           /*tl_params=*/&params);
    return f; });

  size_t num_classes = 0;
  if (classification)
  {
    auto const rc = TreeliteQueryNumClass(/*handle=*/*tl_handle.get(),
                                          /*out=*/&num_classes);
    if (rc < 0)
    {
      return -1;
    }

    // Treelite returns 1 as number of classes for binary classification.
    num_classes = std::max(num_classes, size_t(2));
  }

  size_t num_features = 0;
  {
    auto const rc = TreeliteQueryNumFeature(/*handle=*/*tl_handle.get(),
                                            /*out=*/&num_features);
    if (rc < 0)
    {
      return -1;
    }
  }

  auto model = std::make_unique<FILModel>(
      /*handle=*/std::move(handle),
      std::move(forest),
      num_classes,
      num_features);

  *out = static_cast<FILModelHandle>(model.release());

  return 0;
}

__host__ int FILModelFree(
    FILModelHandle handle)
{
  delete static_cast<FILModel *>(handle);
  return 0;
}

__host__ int FILGetNumClasses(
    FILModelHandle model,
    size_t *out)
{
  auto const model_xptr = static_cast<FILModel const *>(model);
  *out = model_xptr->numClasses_;
  return 0;
}

__host__ int FILPredict(
    FILModelHandle model,
    const float *x,
    size_t num_row,
    bool output_class_probabilities,
    float *out)
{

  auto const fil_model = static_cast<FILModel const *>(model);

  if (output_class_probabilities && fil_model->numClasses_ == 0)
  {
    return -1;
  }

  auto &handle = *(fil_model->handle_);

  auto output_size = output_class_probabilities
                         ? fil_model->numClasses_ * num_row
                         : num_row;

  const auto feature_size = fil_model->numFeatures_ * num_row;
  // ensemble input data
  thrust::device_vector<float> d_x(feature_size);

  // TODO: async copy
  thrust::copy(
      x,
      x + feature_size,
      d_x.begin());

  // ensemble output
  thrust::device_vector<float>
      d_preds(output_size);

  ML::fil::predict(/*h=*/handle,
                   /*f=*/fil_model->forest_.get(),
                   /*preds=*/d_preds.data().get(),
                   /*data=*/d_x.data().get(),
                   /*num_rows=*/num_row,
                   /*predict_proba=*/output_class_probabilities);

  // TODO: async copy
  thrust::copy(
      d_preds.begin(),
      d_preds.end(),
      out);

  return 0;
}
