#include "async_utils.cuh"
#include "cuda_utils.h"
#include "handle_utils.h"
#include "preprocessor.h"
#include "stream_allocator.h"
#include "cuml4c/dbscan.h"

#include <thrust/async/copy.h>
#include <thrust/device_vector.h>
#include <cuml/cluster/dbscan.hpp>

#include <memory>

__host__ int DbscanFit(
    const float *x,
    size_t num_row,
    size_t num_col,
    int min_pts,
    double eps,
    int metric,
    size_t max_bytes_per_batch,
    int verbosity,
    int *out)
{

    const size_t value_length = num_row * num_col;

    auto stream_view = cuml4c::stream_allocator::getOrCreateStream();
    raft::handle_t handle;
    cuml4c::handle_utils::initializeHandle(handle, stream_view.value());

    // dbscan input data
    thrust::device_vector<float> d_src_data(value_length);

    // dbscan output data
    thrust::device_vector<int> d_labels(num_row);

    // TODO: async copy
    thrust::copy(
        x,
        x + value_length,
        d_src_data.begin());

    ML::Dbscan::fit(handle, /*input=*/d_src_data.data().get(),
                    /*n_rows=*/num_row,
                    /*n_cols=*/num_col,
                    eps,
                    min_pts,
                    /*metric=*/static_cast<raft::distance::DistanceType>(metric),
                    /*labels=*/d_labels.data().get(),
                    /*core_sample_indices=*/nullptr,
                    max_bytes_per_batch,
                    /*verbosity=*/verbosity,
                    /*opg=*/false);

    // TODO: async copy
    thrust::copy(
        d_labels.begin(),
        d_labels.end(),
        out);

    return 0;
}
