#include "device_allocator.h"

#include <raft/mr/device/allocator.hpp>

namespace
{

    auto const kDefaultDeviceAllocator =
        std::make_shared<raft::mr::device::default_allocator>();

} // namespace

namespace cuml4c
{

    __host__ std::shared_ptr<raft::mr::device::allocator> getDeviceAllocator()
    {
        return kDefaultDeviceAllocator;
    }

} // namespace cuml4c
